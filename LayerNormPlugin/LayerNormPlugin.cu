#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
 
#include "LayerNormPlugin.h"

using namespace nvinfer1;

PluginFieldCollection LayerNormPluginCreator::fc_{};
std::vector<PluginField> LayerNormPluginCreator::attr_;

__global__ void layerNormKernel(float *pInput, float *pOutput)
{
    const int tx = threadIdx.x, index = blockIdx.x * 768 + threadIdx.x;

    __shared__ float temp[256];

    float value0 = pInput[index];
    float value1 = pInput[index + 256];
    float value2 = pInput[index + 512];

    temp[tx] = value0 + value1 + value2;
    __syncthreads();

    for (int stride = 128; stride >= 1; stride /= 2)
    {
        if (tx < stride)
        {
            temp[tx] += temp[tx + stride];
        }
        __syncthreads();
    }
    float mean = temp[0] / 768;
    __syncthreads();

    temp[tx] = (value0 - mean) * (value0 - mean) + (value1 - mean) * (value1 - mean) + (value2 - mean) * (value2 - mean);
    __syncthreads();

    for (int stride = 128; stride >= 1; stride /= 2)
    {
        if (tx < stride)
        {
            temp[tx] += temp[tx + stride];
        }
        __syncthreads();
    }
    float var = temp[0] / 768;

    pOutput[index]       = (value0 - mean) * rsqrtf(var + 6e-6);
    pOutput[index + 256] = (value1 - mean) * rsqrtf(var + 6e-6);
    pOutput[index + 512] = (value2 - mean) * rsqrtf(var + 6e-6);
}

int32_t LayerNormPlugin::enqueue(const PluginTensorDesc* inputDesc, const PluginTensorDesc* outputDesc, const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept
{
    const int nBlock = inputDesc[0].dims.d[0] * inputDesc[0].dims.d[1];

    layerNormKernel <<<nBlock, 256, 0, stream>>>((float *)inputs[0], (float *)outputs[0]);
    return 0;
}

REGISTER_TENSORRT_PLUGIN(LayerNormPluginCreator);

